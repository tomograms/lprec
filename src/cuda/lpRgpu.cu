#include "hip/hip_runtime.h"
#include "lpRgpu.cuh"
#include "main_kernels.cuh"
#include "simple_kernels.cuh"


//init global parameters
lpRgpu::lpRgpu(size_t params, int gpu)
{  
	hipSetDevice(gpu);
	readGlobalParametersArr((float*)params);
	err = hipMalloc((void **)&derho, Ntheta*Nrho*sizeof(float)); if (err!=0) callErr(hipGetErrorString(err));
	err = hipMalloc((void **)&dfl, Nslices*Ntheta*Nrho*sizeof(float)); if (err!=0) callErr(hipGetErrorString(err));
	err = hipMalloc((void **)&dflc, Nslices*Ntheta_R2C*Nrho*sizeof(float2)); if (err!=0) callErr(hipGetErrorString(err));

	//init rho space
	hipMemcpy(derho,erho,Ntheta*Nrho*sizeof(float),hipMemcpyHostToDevice);

	hipChannelFormatDesc texf_desc = hipCreateChannelDesc<float>();	
	hipExtent volumeSize = make_hipExtent(Ntheta,Nrho,Nslices);

	err = hipMalloc3DArray(&dfla, &texf_desc,volumeSize,hipArrayLayered); if (err!=0) callErr(hipGetErrorString(err));
	//texfl.addressMode[0] = hipAddressModeWrap;
	//texfl.addressMode[1] = hipAddressModeWrap;
	//texfl.filterMode = hipFilterModeLinear;
	//texfl.normalized  = true;
	//hipBindTextureToArray(texfl, dfla,texf_desc);

	// Specify texture
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = dfla;

	// Specify texture object parameters
	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 1;

	// Create texture object
	texflObj = 0;
	hipCreateTextureObject(&texflObj, &resDesc, &texDesc, NULL);

	texfObj = 0;
	texRObj = 0;

	//fft plans for Nslices slices
	hipfftResult res1,res2;
	int ffts[] = {Nrho,Ntheta};
	int idist = Nrho*Ntheta;int odist = Nrho*(Ntheta/2+1);
	int inembed[] = {Nrho, Ntheta};int onembed[] = {Nrho, Ntheta/2+1};
	res1 = hipfftPlanMany(&plan_forward, 2, ffts, inembed, 1, idist, onembed, 1, odist, HIPFFT_R2C, Nslices); if (res1!=0) {char errs[16];sprintf(errs,"fwd hipfftPlanMany error %d",res1);callErr(errs);}
	res2 = hipfftPlanMany(&plan_inverse, 2, ffts, onembed, 1, odist, inembed, 1, idist, HIPFFT_C2R, Nslices); if (res2!=0) {char errs[16];sprintf(errs,"inv hipfftPlanMany error %d",res1);callErr(errs);}

	err = hipMalloc((void **)&dR, Nslices*Nproj*N*sizeof(float));	if (err!=0) callErr(hipGetErrorString(err));
	err = hipMalloc((void **)&df, Nslices*N*N*sizeof(float));	if (err!=0) callErr(hipGetErrorString(err));
	dfZfwd = NULL;//for checking if fwd is created
}

lpRgpu::~lpRgpu()
{
	delete[] erho;
	//free gpu memory
	hipFree(derho);
	hipFree(dfl);
	hipFree(dflc);
	//hipUnbindTexture(texfl);
	// Destroy texture object
	hipDestroyTextureObject(texflObj);
	hipFreeArray(dfla);	
	hipfftDestroy(plan_forward);
	hipfftDestroy(plan_inverse);
	hipFree(df);
	hipFree(dR);

	//delete parameters for fwd and adj transform if they are initialized
	if(dfZfwd) deleteFwd();
	deleteAdj();
}

//init parameters for forward (Radon) tranform 
void lpRgpu::initFwd(size_t paramsi, size_t paramsf, int gpu)
{
	hipSetDevice(gpu);
	fgs = new fwdgrids(Nspan);
	readFwdParametersArr((int*)paramsi, (float*)paramsf);

	fgs->initgpu();
	err = hipMalloc((void **)&dfZfwd, Ntheta_R2C*Nrho*sizeof(float2)); if (err!=0) callErr(hipGetErrorString(err));
	err = hipMalloc((void **)&dtmpf, Nslices*N*N*sizeof(float)); if (err!=0) callErr(hipGetErrorString(err));//delete to do

	//copy Fourier transform of Z	
	hipMemcpy(dfZfwd,fZfwd,Ntheta_R2C*Nrho*sizeof(float2),hipMemcpyHostToDevice);

	hipChannelFormatDesc texf_desc = hipCreateChannelDesc<float>();
	hipExtent volumeSize = make_hipExtent(N,N,Nslices); 
	err = hipMalloc3DArray(&dfa, &texf_desc, volumeSize,hipArrayLayered); if (err!=0) callErr(hipGetErrorString(err));
	//texf.addressMode[0] = hipAddressModeWrap;
	//texf.addressMode[1] = hipAddressModeWrap;	
	//texf.filterMode = hipFilterModeLinear;
	//texf.normalized = true;
	//hipBindTextureToArray(texf, dfa,texf_desc);

	// Specify texture
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = dfa;

	// Specify texture object parameters
	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 1;

	// Create texture object
	texfObj = 0;
	hipCreateTextureObject(&texfObj, &resDesc, &texDesc, NULL);
	
	//init result with zeros
	err = hipMemset(dR, 0, Nslices*Nproj*N*sizeof(float)); if (err!=0) callErr(hipGetErrorString(err));
}
void lpRgpu::deleteFwd()
{
	delete[] fZfwd;
	hipFree(dtmpf);
	//hipUnbindTexture(texf);
	// Destroy texture object
	hipDestroyTextureObject(texfObj);	
	hipFreeArray(dfa);
	hipFree(dfZfwd);
	delete fgs;
	err = hipGetLastError(); if(err!=0) callErr(hipGetErrorString(err));
}

//init parameters for adjoint tranform (back-projection)
void lpRgpu::initAdj(size_t paramsi, size_t paramsf, int gpu)
{
	hipSetDevice(gpu);
	ags = new adjgrids(Nspan);
	readAdjParametersArr((int*)paramsi, (float*)paramsf);
	ags->initgpu();

	err = hipMalloc((void **)&dfZadj, Ntheta_R2C*Nrho*sizeof(float2)); if (err!=0) callErr(hipGetErrorString(err));
	err = hipMalloc((void **)&dtmpR, Nslices*N*Nproj*sizeof(float)); if (err!=0) callErr(hipGetErrorString(err));//delete to do

	//copy Fourier transform of adj Z
	hipMemcpy(dfZadj,fZadj,Ntheta_R2C*Nrho*sizeof(float2),hipMemcpyHostToDevice);

	hipChannelFormatDesc texf_desc = hipCreateChannelDesc<float>();	
	hipExtent volumeSize = make_hipExtent(N,Nproj,Nslices); 
	err = hipMalloc3DArray(&dRa, &texf_desc, volumeSize,hipArrayLayered); if (err!=0) callErr(hipGetErrorString(err));
	//texR.addressMode[0] = hipAddressModeWrap;
	//texR.addressMode[1] = hipAddressModeWrap;
	//texR.filterMode = hipFilterModeLinear;
	//texR.normalized = true;
	//hipBindTextureToArray(texR, dRa,texf_desc);
	
	// Specify texture
	struct hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeArray;
	resDesc.res.array.array = dRa;

	// Specify texture object parameters
	struct hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.addressMode[0] = hipAddressModeWrap;
	texDesc.addressMode[1] = hipAddressModeWrap;
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.readMode = hipReadModeElementType;
	texDesc.normalizedCoords = 1;

	// Create texture object
	texRObj = 0;
	hipCreateTextureObject(&texRObj, &resDesc, &texDesc, NULL);

	//init result with zeros
	err = hipMemset(df, 0, Nslices*N*N*sizeof(float)); if (err!=0) callErr(hipGetErrorString(err));

	//init filter
	if (filter)
	{
		int osfilter = 4;
		err = hipMalloc((void **)&dfilter, N*osfilter*sizeof(float)); if (err!=0) callErr(hipGetErrorString(err));
		hipMemcpy(dfilter, filter,N*osfilter*sizeof(float),hipMemcpyDefault); 	
		hipfftPlan1d(&plan_f_forward,N*osfilter,HIPFFT_C2C,Nproj);
		hipfftPlan1d(&plan_f_inverse,N*osfilter,HIPFFT_C2C,Nproj);
		err = hipMalloc((void **)&dRc, Nproj*N*osfilter*sizeof(float2)); if (err!=0) callErr(hipGetErrorString(err));
	}
}
void lpRgpu::deleteAdj()
{
	delete[] fZadj;
	hipFree(dtmpR);
	hipFree(dfZadj);
	//hipUnbindTexture(texR);
	// Destroy texture object
	hipDestroyTextureObject(texRObj);
	hipFreeArray(dRa);
	delete ags;

	if(filter)
	{
		delete[] filter;
		hipFree(dfilter);
		hipFree(dRc);
		hipfftDestroy(plan_f_forward);
		hipfftDestroy(plan_f_inverse);
	}	
}

hipError_t copy3DDeviceToArray(hipArray* dfa, float* df, hipExtent ext)
{
	hipMemcpy3DParms param = { 0 };
	param.srcPtr   = make_hipPitchedPtr((void*)df, ext.width*sizeof(float), ext.width, ext.height);
	param.dstArray = dfa;
	param.kind = hipMemcpyDeviceToDevice;
	param.extent = ext;
	return hipMemcpy3D(&param);
}

hipError_t copy3Dshifted(float *dst, int dstx,int dsty, hipExtent dstext, float* src, int srcx, int srcy, hipExtent srcext, hipExtent copyext)
{
	hipMemcpy3DParms param = { 0 };
	param.srcPtr = make_hipPitchedPtr(&src[srcy*srcext.width+srcx], srcext.width*sizeof(float), srcext.width, srcext.height);
	param.dstPtr = make_hipPitchedPtr(&dst[dsty*dstext.width+dstx], dstext.width*sizeof(float), dstext.width, dstext.height);
	param.kind = hipMemcpyDefault;
	copyext.width*= sizeof(float);
	param.extent = copyext;
	return hipMemcpy3D(&param);
}

void copy3Dstep(float *dst, float* src, int stepy,int N, int Nproj, int Nslices, bool flg)
{
	uint GS31 = (uint)ceil(N/(float)MBS21);uint GS32 = (uint)ceil(Nproj/stepy/(float)MBS22);uint GS33 = (uint)ceil(Nslices/(float)MBS33);
	dim3 dimBlock(MBS31,MBS32,MBS33);dim3 dimGrid(GS31,GS32,GS33);
	if(flg)
		stepangles<<<dimGrid,dimBlock>>>(dst,src,stepy,1,N,Nproj/stepy,Nslices);	
	else
		stepangles<<<dimGrid,dimBlock>>>(dst,src,1,stepy,N,Nproj/stepy,Nslices);	
}

void lpRgpu::execFwdManyPtr(size_t Rptr, size_t fptr, int Nslices0, int gpu)
{
	hipSetDevice(gpu);
	hipMemset(df,0,N*N*Nslices*sizeof(float));
	hipMemset(dtmpR,0,Nproj*N*Nslices*sizeof(float));
	copy3Dshifted(df,N/2-N0/2,N/2-N0/2,make_hipExtent(N,N,Nslices),(float*)fptr,0,0,make_hipExtent(N0, N0, Nslices0),make_hipExtent(N0,N0,Nslices0));
	execFwd();
	copy3Dstep(dtmpR, dR, osangles, N, Nproj, Nslices, 0);
    copy3Dshifted((float*)Rptr,0,0,make_hipExtent(N0,Nproj/osangles,Nslices0),dtmpR,N/2-cor,0,make_hipExtent(N, Nproj/osangles, Nslices),make_hipExtent(N0,Nproj/osangles,Nslices0));
}

//compute back-projection for several slices
void lpRgpu::execAdjManyPtr(size_t fptr, size_t Rptr, int Nslices0, int gpu)
{
	hipSetDevice(gpu);
	hipMemset(dR,0,Nproj*N*Nslices*sizeof(float));
	hipMemset(dtmpR,0,Nproj*N*Nslices*sizeof(float));
	copy3Dshifted(dtmpR,N/2-cor,0,make_hipExtent(N, Nproj/osangles, Nslices),(float*)Rptr,0,0,make_hipExtent(N0,Nproj/osangles,Nslices0),make_hipExtent(N0,Nproj/osangles,Nslices0));
	copy3Dstep(dR, dtmpR, osangles, N, Nproj, Nslices, 1);
	padding(N0);
	applyFilter();
	execAdj();
    copy3Dshifted((float*)fptr,0,0,make_hipExtent(N0, N0, Nslices0),df,N/2-N0/2,N/2-N0/2,make_hipExtent(N,N,Nslices),make_hipExtent(N0,N0,Nslices0));
}


//padding
void lpRgpu::padding(int N_)
{
	uint GS31 = (uint)ceil(N/(float)MBS21);uint GS32 = (uint)ceil(Nproj/(float)MBS22);uint GS33 = (uint)ceil(Nslices/(float)MBS33);
    dim3 dimBlock(MBS31,MBS32,MBS33);dim3 dimGrid(GS31,GS32,GS33);
    padker<<<dimGrid,dimBlock>>>(dR,N/2-cor,N/2+N_-cor-1,N,Nproj,Nslices);
}

//prefilter to compensate amplitudes in cubic interpolation
void lpRgpu::prefilter2D(float *df, float* dtmpf, uint width, uint height)
{
	//transpose for optimal cache usage
	uint GS31 = (uint)ceil(width/(float)MBS31);uint GS32 = (uint)ceil(height/(float)MBS32);uint GS33 = (uint)ceil(Nslices/(float)MBS33);
	dim3 dimBlock(MBS31,MBS32,MBS33);dim3 dimGrid(GS31,GS32,GS33);
	transpose<<<dimGrid,dimBlock>>>(dtmpf, df,width, height,Nslices);

	//compensate in samples for x direction
	uint GS41 = (uint)ceil(height/(float)MBS41);uint GS42 = (uint)ceil(Nslices/(float)MBS42); 
	dim3 dimBlock1(MBS41,MBS42);dim3 dimGrid1(GS41,GS42);
	SamplesToCoefficients2DY<<<dimGrid1, dimBlock1>>>(dtmpf,height*sizeof(float),height, width,Nslices);

	//transpose back
	GS31 = (uint)ceil(height/(float)MBS31);GS32 = (uint)ceil(width/(float)MBS32);GS33 = (uint)ceil(Nslices/(float)MBS33);
	dim3 dimBlock2(MBS31,MBS32,MBS33);dim3 dimGrid2(GS31,GS32,GS33);
	transpose<<<dimGrid2,dimBlock2>>>(df,dtmpf,height, width,Nslices);

	//compensate in samples for y direction
	GS41 = (uint)ceil(width/(float)MBS41);GS42 = (uint)ceil(Nslices/(float)MBS42); 
	dim3 dimBlock3(MBS41,MBS42);dim3 dimGrid3(GS41,GS42);	
	SamplesToCoefficients2DY<<<dimGrid3, dimBlock3>>>(df,width*sizeof(float),width,height,Nslices);
}

//compute Radon transform in log-polar coordinates
void lpRgpu::execFwd()
{
	err = hipMemset(dtmpf, 0, Nslices*N*N*sizeof(float)); if (err!=0) callErr(hipGetErrorString(err));
	err = hipMemset(dR, 0, Nslices*Nproj*N*sizeof(float)); if (err!=0) callErr(hipGetErrorString(err));
	//compensation for cubic interpolation
	if(interp_type) prefilter2D(df,dtmpf,N,N);

	//init gpu array with binded texture
	copy3DDeviceToArray(dfa,df,make_hipExtent(N, N, Nslices));

	//CUDA block and grid sizes
	dim3 dimBlock(MBS31,MBS32,MBS33);
	uint GS31, GS32, GS33;

	for(int k = 0;k<Nspan;k++)
	{
		err = hipMemset(dfl, 0, Nslices*Ntheta*Nrho*sizeof(float)); if (err!=0) callErr(hipGetErrorString(err)); 

		//interp Cartesian to log-polar grid
		GS31 = (uint)ceil(ceil(sqrtf((float)fgs->Ncidsfwd))/(float)MBS31);GS32 = (uint)ceil(ceil(sqrtf((float)fgs->Ncidsfwd))/(float)MBS32);GS33 = (uint)ceil(Nslices/(float)MBS33);dim3 dimGrid(GS31,GS32,GS33);
		interp<<<dimGrid, dimBlock>>>(0+interp_type*3,texfObj,texRObj,texflObj,dfl,fgs->dlp2C1[k],fgs->dlp2C2[k],MBS31*GS31,fgs->Ncidsfwd,N,N,Nslices,fgs->dcidsfwd,Ntheta*Nrho);
		
		//multiplication e^{\rho}
		GS31 = (uint)ceil(Ntheta/(float)MBS31);GS32 = (uint)ceil(Nrho/(float)MBS32);GS33 = (uint)ceil(Nslices/(float)MBS33);dim3 dimGrid1(GS31,GS32,GS33);
		mulexp<<<dimGrid1, dimBlock>>>(dfl,derho,Ntheta,Nrho, Nslices);

		//forward FFT
		hipfftExecR2C(plan_forward, (hipfftReal*)dfl,(hipfftComplex*)dflc);

		//multiplication by fZ
		GS31 = (uint)ceil(Ntheta_R2C/(float)MBS31);GS32 = (uint)ceil(Nrho/(float)MBS32);GS33 = (uint)ceil(Nslices/(float)MBS33);dim3 dimGrid2(GS31,GS32,GS33);
		mul<<<dimGrid2, dimBlock>>>(1/(float)(Ntheta*Nrho),dflc,dfZfwd,Ntheta_R2C,Nrho,Nslices);

		//inverse FFT
		hipfftExecC2R(plan_inverse,(hipfftComplex*)dflc,(hipfftReal*)dfl);

		//init gpu array with binded texture
		copy3DDeviceToArray(dfla,dfl,make_hipExtent(Ntheta, Nrho, Nslices));

		//interp log-polar to polar grid
		GS31 = (uint)ceil(ceil(sqrtf((float)fgs->Npids[k]))/(float)MBS31);GS32 = (uint)ceil(ceil(sqrtf((float)fgs->Npids[k]))/(float)MBS32);GS33 = (uint)ceil(Nslices/(float)MBS33);dim3 dimGrid3(GS31,GS32,GS33);
		interp<<<dimGrid3, dimBlock>>>(2+interp_type*3,texfObj,texRObj,texflObj,dR,fgs->dp2lp1[k],fgs->dp2lp2[k],MBS31*GS31,fgs->Npids[k],Ntheta,Nrho,Nslices,fgs->dpids[k],Nproj*N);
	}
}

//compute back-projection in log-polar coordinates
void lpRgpu::execAdj()
{
	hipMemset(dtmpR, 0, Nslices*Nproj*N*sizeof(float)); 
	hipMemset(df, 0, Nslices*N*N*sizeof(float)); 
	//compensation for cubic interpolation

	if(interp_type) prefilter2D(dR,dtmpR,N,Nproj);
	//init gpu array with binded texture
	copy3DDeviceToArray(dRa,dR,make_hipExtent(N, Nproj, Nslices));

	//CUDA block and grid sizes
	dim3 dimBlock(MBS31,MBS32,MBS33);
	uint GS31, GS32, GS33;
	for(int k = 0;k<Nspan;k++)
	{   
		hipMemset(dfl, 0, Nslices*Ntheta*Nrho*sizeof(float)); 
		//interp from polar to log-polar grid
		GS31 = (uint)ceil(ceil(sqrt(ags->Nlpidsadj))/(float)MBS31); GS32 = (uint)ceil(ceil(sqrt(ags->Nlpidsadj))/(float)MBS32);GS33 = (uint)ceil(Nslices/(float)MBS33);dim3 dimGrid(GS31,GS32,GS33);
		interp<<<dimGrid, dimBlock>>>(1+interp_type*3,texfObj,texRObj,texflObj,dfl,ags->dlp2p2[k],ags->dlp2p1[k],MBS31*GS31,ags->Nlpidsadj,N,Nproj,Nslices,ags->dlpidsadj,Ntheta*Nrho);

		//interp from polar to log-polar grid additional points
		GS31 = (uint)ceil(ceil(sqrt(ags->Nwids))/(float)MBS31); GS32 = (uint)ceil(ceil(sqrt(ags->Nwids))/(float)MBS32);GS33 = (uint)ceil(Nslices/(float)MBS33);dim3 dimGrid4(GS31,GS32,GS33);
		interp<<<dimGrid4, dimBlock>>>(1+interp_type*3,texfObj,texRObj,texflObj,dfl,ags->dlp2p2w[k],ags->dlp2p1w[k],MBS31*GS31,ags->Nwids,N,Nproj,Nslices,ags->dwids,Ntheta*Nrho);

		//Forward FFT
		hipfftExecR2C(plan_forward, (hipfftReal*)dfl,(hipfftComplex*)dflc);

		//multiplication by adjoint fZ
		GS31 = (uint)ceil(Ntheta_R2C/(float)MBS31); GS32 = (uint)ceil(Nrho/(float)MBS32);GS33 = (uint)ceil(Nslices/(float)MBS33);dim3 dimGrid2(GS31,GS32,GS33);
		mul<<<dimGrid2, dimBlock>>>(1/(float)(Ntheta*Nrho),dflc,dfZadj,Ntheta_R2C,Nrho,Nslices);

		//Inverse FFT
		hipfftExecC2R(plan_inverse,(hipfftComplex*)dflc,(hipfftReal*)dfl);

		//init gpu array with binded texture
		copy3DDeviceToArray(dfla,dfl,make_hipExtent(Ntheta, Nrho, Nslices));

		//interp from log-polar to Cartesian grid
		GS31 = (uint)ceil(ceil(sqrt(ags->Ncidsadj))/(float)MBS31); GS32 = (uint)ceil(ceil(sqrt(ags->Ncidsadj))/(float)MBS32);GS33 = (uint)ceil(Nslices/(float)MBS33);dim3 dimGrid3(GS31,GS32,GS33);
		interp<<<dimGrid3, dimBlock>>>(2+interp_type*3,texfObj,texRObj,texflObj,df,ags->dC2lp1[k],ags->dC2lp2[k],MBS31*GS31,ags->Ncidsadj,Ntheta,Nrho,Nslices,ags->dcidsadj,N*N);
	}
}

//apply filter in frequency
void lpRgpu::applyFilter()
{
	if (!filter) return;
	
	dim3 dimBlock(MBS21,MBS22);
	uint GS21, GS22;
	int osfilter = 4;
	for(int ij = 0;ij<Nslices;ij++)
	{
		hipMemset(dRc, 0, 2*Nproj*N*osfilter*sizeof(float));
		
		//copy to complex array
		GS21 = ceil(N/(float)MBS21);GS22 = ceil(Nproj/(float)MBS22);dim3 dimGrid1(GS21,GS22);	
		copyc<<<dimGrid1, dimBlock>>>(&dR[N*Nproj*ij],dRc,N,Nproj,osfilter);

		//fftshift 
		GS21 = ceil(N*osfilter/(float)MBS21);GS22 = ceil(Nproj/(float)MBS22);dim3 dimGrid2(GS21,GS22);
		fftshift<<<dimGrid2, dimBlock>>>(dRc,N*osfilter,Nproj);

		//forward fft	
		hipfftExecC2C(plan_f_forward,dRc,dRc,HIPFFT_FORWARD);
	
		//fftshift
		GS21 = ceil(N*osfilter/(float)MBS21);GS22 = ceil(Nproj/(float)MBS22);dim3 dimGrid3(GS21,GS22);
		fftshift<<<dimGrid3, dimBlock>>>(dRc,N*osfilter,Nproj);

		//mulfilter
		GS21 = ceil(N*osfilter/(float)MBS21);GS22 = ceil(Nproj/(float)MBS22);dim3 dimGrid4(GS21,GS22);
		mulfilter<<<dimGrid4, dimBlock>>>(dRc,dfilter,N*osfilter,Nproj);

		//fftshift
		GS21 = ceil(N*osfilter/(float)MBS21);GS22 = ceil(Nproj/(float)MBS22);dim3 dimGrid5(GS21,GS22);
		fftshift<<<dimGrid5, dimBlock>>>(dRc,N*osfilter,Nproj);
	
		//adjoint fft	
		hipfftExecC2C(plan_f_inverse,dRc,dRc,HIPFFT_BACKWARD);
	
		//fftshift
		GS21 = ceil(N*osfilter/(float)MBS21);GS22 = ceil(Nproj/(float)MBS22);dim3 dimGrid6(GS21,GS22);
		fftshift<<<dimGrid6, dimBlock>>>(dRc,N*osfilter,Nproj);

		//copy from complex array
		GS21 = ceil(N/(float)MBS21);GS22 = ceil(Nproj/(float)MBS22);dim3 dimGrid7(GS21,GS22);	
		copycback<<<dimGrid7, dimBlock>>>(&dR[N*Nproj*ij],dRc,N,Nproj,osfilter);

		//mul const
		GS21 = ceil(N/(float)MBS21);GS22 = ceil(Nproj/(float)MBS22);dim3 dimGrid9(GS21,GS22);
		mulconst<<<dimGrid9, dimBlock>>>(&dR[Nproj*N*ij],1/(float)(osfilter*N), N, Nproj);
	}
}

